#include "utils.cuh"

void cuda_log(hipError_t err)
{
    LAST_CUDA_ERROR = err;
    LAST_CUDA_ERROR_DESC = hipGetErrorString(err);
}

matrix* transferMatrixToDevice(matrix* h_m)
{
    matrix* d_m;
    unsigned char* h_arr = h_m->get_arr_interlaced();
    unsigned char* d_arr;

    cuda_log(hipMalloc(&d_m,  sizeof(matrix)));
    cuda_log(hipMalloc(&d_arr, h_m->size_interlaced()));
    cuda_log(hipMemcpy(d_arr, h_arr, h_m->size_interlaced(), hipMemcpyHostToDevice));
    h_m->set_arr_interlaced(d_arr);

    cuda_log(hipMemcpy(d_m, h_m, sizeof(matrix), hipMemcpyHostToDevice));

    h_m->set_arr_interlaced(h_arr);

    return d_m;
}

void transferMatrixDataToHost(matrix* h_m, matrix* d_m, bool do_free)
{
    unsigned char* h_arr = h_m->get_arr_interlaced();

    cuda_log(hipMemcpy(h_m, d_m, sizeof(matrix), hipMemcpyDeviceToHost));
    cuda_log(hipMemcpy(h_arr, h_m->get_arr_interlaced(), h_m->size_interlaced(), hipMemcpyDeviceToHost));

    if (do_free)
    {
        hipFree(h_m->get_arr_interlaced());
        hipFree(d_m);
    }

    h_m->set_arr_interlaced(h_arr);
}
