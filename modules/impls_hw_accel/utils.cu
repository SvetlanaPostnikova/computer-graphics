#include "utils.cuh"

__host__ __device__ void cuda_log_detailed(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        const char* desc = hipGetErrorString(err);
        printf("[%s:%d] CUDA ERROR %d: %s\n", file, line, err, desc);
    }
}

__host__ __device__ void cuda_log_detailed(hipblasStatus_t err, const char *file, int line)
{
    if (err != HIPBLAS_STATUS_SUCCESS)
    {
        printf("[%s:%d] CUBLAS ERROR %d\n", file, line, err);
    }
}

__host__ __device__ void cuda_log_detailed(nvjpegStatus_t status, const char *file, int line)
{
    if (status != NVJPEG_STATUS_SUCCESS)
    {
        printf("[%s:%d] NVJPEG ERROR %d\n", file, line, status);
    }
}

void transferMatrixToDevice(matrix* d_m, unsigned char* d_arr_interlaced, matrix* h_m)
{
    unsigned char* h_arr = h_m->get_arr_interlaced();

    cuda_log(hipMemcpy(d_arr_interlaced, h_arr, h_m->size_interlaced(), hipMemcpyHostToDevice));
    h_m->set_arr_interlaced(d_arr_interlaced);

    cuda_log(hipMemcpy(d_m, h_m, sizeof(matrix), hipMemcpyHostToDevice));

    h_m->set_arr_interlaced(h_arr);
}

void transferMatrixDataToHost(matrix* h_m, matrix* d_m, bool do_free)
{
    unsigned char* h_arr = h_m->get_arr_interlaced();

    cuda_log(hipMemcpy(h_m, d_m, sizeof(matrix), hipMemcpyDeviceToHost));
    cuda_log(hipMemcpy(h_arr, h_m->get_arr_interlaced(), h_m->size_interlaced(), hipMemcpyDeviceToHost));

    if (do_free)
    {
        cuda_log(hipFree(h_m->get_arr_interlaced()));
        cuda_log(hipFree(d_m));
    }

    h_m->set_arr_interlaced(h_arr);
}
