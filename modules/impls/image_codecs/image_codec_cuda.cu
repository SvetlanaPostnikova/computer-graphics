#include "image_codec.h"
#include "nvjpeg.h"
#include <fstream>

hipStream_t stream;
nvjpegHandle_t nv_handle;

nvjpegJpegState_t nvjpeg_decoder_state;

nvjpegEncoderState_t nv_enc_state;
nvjpegEncoderParams_t nv_enc_params;

/// @brief for debug
nvjpegStatus_t last_status = (nvjpegStatus_t)-1;
hipError_t last_error = (hipError_t)-1;
std::string last_error_desc = "";

void cuda_log(nvjpegStatus_t status)
{
    last_status = status;
}

void cuda_log(hipError_t status)
{
    last_error = status;
    last_error_desc = hipGetErrorString(status);
}

image_codec::image_codec()
{
    //THREAD SAFE
    //cuda stream that stores order of operations on GPU
    cuda_log(hipStreamCreate(&stream));
    //library handle
    cuda_log(nvjpegCreateSimple(&nv_handle));

    //NOT THREAD SAFE
    //nvjpeg encoding
    cuda_log(nvjpegEncoderStateCreate(nv_handle, &nv_enc_state, stream));
    cuda_log(nvjpegEncoderParamsCreate(nv_handle, &nv_enc_params, stream));

    // set the highest quality
    cuda_log(nvjpegEncoderParamsSetQuality(nv_enc_params, 100, stream));

    //use the best type of JPEG encoding
    cuda_log(nvjpegEncoderParamsSetEncoding(nv_enc_params, nvjpegJpegEncoding_t::NVJPEG_ENCODING_LOSSLESS_HUFFMAN, stream));

    //nvjpeg decoding
    cuda_log(nvjpegJpegStateCreate(nv_handle, &nvjpeg_decoder_state));
}

void image_codec::encode(std::vector<unsigned char>* img_source, matrix* img_matrix, ImageColorScheme colorScheme, unsigned bit_depth)
{
    // code taken from example: https://docs.nvidia.com/cuda/nvjpeg/index.html#nvjpeg-encode-examples

    nvjpegImage_t nv_image;
    //Pitch represents bytes per row
    size_t pitch_0_size = img_matrix->width;

    if (colorScheme == ImageColorScheme::IMAGE_RGB)
    {
        // This has to be done, default params are not sufficient
        // source: https://stackoverflow.com/questions/65929613/nvjpeg-encode-packed-bgr
        cuda_log(nvjpegEncoderParamsSetSamplingFactors(nv_enc_params, NVJPEG_CSS_444, stream));

        pitch_0_size *= 3;
    }
    else
    {
        cuda_log(nvjpegEncoderParamsSetSamplingFactors(nv_enc_params, NVJPEG_CSS_GRAY, stream));
    }

    // Fill nv_image with image data, by copying data from matrix to GPU
    // docs about nv_image: https://docs.nvidia.com/cuda/nvjpeg/index.html#nvjpeg-encode-examples
    cuda_log(hipMalloc((void **)&(nv_image.channel[0]), pitch_0_size * img_matrix->height));
    cuda_log(hipMemcpy(nv_image.channel[0], img_matrix->array.data(), pitch_0_size * img_matrix->height, hipMemcpyHostToDevice));
    
    nv_image.pitch[0] = pitch_0_size;

    // Compress image
    if (colorScheme == ImageColorScheme::IMAGE_RGB)
    {
        cuda_log(nvjpegEncodeImage(nv_handle, nv_enc_state, nv_enc_params,
            &nv_image, nvjpegInputFormat_t::NVJPEG_INPUT_RGBI, img_matrix->width, img_matrix->height, stream));   
    }
    else
    {
        cuda_log(nvjpegEncodeYUV(nv_handle, nv_enc_state, nv_enc_params,
            &nv_image, nvjpegChromaSubsampling_t::NVJPEG_CSS_GRAY, img_matrix->width, img_matrix->height, stream));
    }

    // get compressed stream size
    size_t length = 0;
    cuda_log(nvjpegEncodeRetrieveBitstream(nv_handle, nv_enc_state, NULL, &length, stream));
    // get stream itself
    cuda_log(hipStreamSynchronize(stream));
    img_source->clear();
    img_source->resize(length);
    cuda_log(nvjpegEncodeRetrieveBitstream(nv_handle, nv_enc_state, img_source->data(), &length, 0));

    cuda_log(hipStreamSynchronize(stream));

    //clean up
    cuda_log(hipFree(nv_image.channel[0]));
}

bool is_interleaved(nvjpegOutputFormat_t)
{
    return true;
}

void image_codec::decode(std::vector<unsigned char>* img_source, matrix* img_matrix, ImageColorScheme colorScheme, unsigned bit_depth)
{
    // Decode, Encoder format
    nvjpegOutputFormat_t oformat = NVJPEG_OUTPUT_RGBI;

    // Image buffers. 
    unsigned char * pBuffer = NULL; 
    
    unsigned char * dpImage = (unsigned char *)img_source->data();
    size_t nSize = img_source->size();
    
    // Retrieve the componenet and size info.
    int nComponent = 0;
    nvjpegChromaSubsampling_t subsampling;
    int widths[NVJPEG_MAX_COMPONENT];
    int heights[NVJPEG_MAX_COMPONENT];

    cuda_log(nvjpegGetImageInfo(nv_handle, dpImage, nSize, &nComponent, &subsampling, widths, heights));

    // image resize
    size_t pitchDesc;

    // device image buffers.
    nvjpegImage_t imgDesc;

    if (is_interleaved(oformat))
    {
        pitchDesc = nComponent * widths[0];
    }
    else
    {
        pitchDesc = 3 * widths[0];
    }

    cuda_log(hipMalloc(&pBuffer, pitchDesc * heights[0]));

    imgDesc.channel[0] = pBuffer;
    imgDesc.channel[1] = pBuffer + widths[0] * heights[0];
    imgDesc.channel[2] = pBuffer + widths[0] * heights[0] * 2;
    imgDesc.pitch[0] = (unsigned int)(is_interleaved(oformat) ? widths[0] * nComponent : widths[0]);
    imgDesc.pitch[1] = (unsigned int)widths[0];
    imgDesc.pitch[2] = (unsigned int)widths[0];

    if (is_interleaved(oformat))
    {
        imgDesc.channel[3] = pBuffer + widths[0] * heights[0] * 3;
        imgDesc.pitch[3] = (unsigned int)widths[0];
    }

    // decode by stages
    cuda_log(nvjpegDecode(nv_handle, nvjpeg_decoder_state, dpImage, nSize, oformat, &imgDesc, NULL));

    img_matrix->array.resize(pitchDesc * heights[0]);
    unsigned char* result = new unsigned char[pitchDesc * heights[0]];

    cuda_log(hipMemcpy(img_matrix->array.data(), pBuffer, pitchDesc * heights[0], hipMemcpyKind::hipMemcpyDeviceToHost));

    img_matrix->height = heights[0];
    img_matrix->width = widths[0];
}

void image_codec::load_image_file(std::vector<unsigned char>* img_buff, std::string image_filepath)
{
    std::ifstream oInputStream(image_filepath, std::ios::in | std::ios::binary | std::ios::ate);
    if(!(oInputStream.is_open()))
    {
        return;
    }

    // Get the size.
    std::streamsize nSize = oInputStream.tellg();
    oInputStream.seekg(0, std::ios::beg);
    
    img_buff->resize(nSize);
    oInputStream.read((char*)img_buff->data(), nSize);

    oInputStream.close();
}
        
void image_codec::save_image_file(std::vector<unsigned char>* img_buff, std::string image_filepath)
{
    std::ofstream output_file(image_filepath+".jpeg", std::ios::out | std::ios::binary);
    output_file.write((char *)img_buff->data(), img_buff->size());
    output_file.close();
}

image_codec::~image_codec()
{
    if (nv_enc_params != nullptr)
    {
        cuda_log(nvjpegEncoderParamsDestroy(nv_enc_params));
        nv_enc_params = nullptr;
    }
    
    if (nv_enc_state != nullptr)
    {
        cuda_log(nvjpegEncoderStateDestroy(nv_enc_state));
        nv_enc_state = nullptr;
    }

    if (nv_handle != nullptr)
    {
        cuda_log(nvjpegDestroy(nv_handle));
        nv_handle = nullptr;
    }

    if (stream != nullptr)
    {
        cuda_log(hipStreamDestroy(stream));
        stream = nullptr;
    }
}
